#include "hip/hip_runtime.h"
#include <vector>
#include <limits>
#include "color.h"
#include "camera.h"
#include "render.h"
#include "ray.h"
#include "vector3.h"
#include "rectangle.h"

static constexpr int CHUNK_SIZE = 32;
static constexpr float FLOAT_INFINITY = std::numeric_limits<float>::max();

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

static __device__ RGBColor cast_ray(Vector3 ray_source, Ray const &ray, Sphere *spheres, int const & spheres_count) {
    float current_distance = INFINITY;
    float hit_sphere = -1;

    for (int sphere_index = 0; sphere_index < spheres_count; sphere_index++) {

        if (spheres[sphere_index].hits_ray(ray)) {
            float distance = Vector3(ray_source, spheres[sphere_index].get_position()).length();
            if (distance < current_distance) {
                current_distance = distance;
                hit_sphere = sphere_index;
            }
        } 
    }

    if (hit_sphere != -1) {
        return RGBColor(1, 0.5f, 1);
    }
    
    return RGBColor(0, 0, 0); // background
}

static __global__ void kernel(int width, int height, RGBColor *img, Sphere *spheres, Camera camera,
                              int spheres_count) {
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    if (tidX > width || tidY > height) {
        return;
    }
    
    Rectangle screen = camera.get_screen();

    
    Vector3 point_on_screen = Vector3(screen.left_top_point.x + tidX * screen.width() / width,
    screen.left_top_point.y - tidY * screen.height() / height, screen.left_top_point.z);
    Vector3 direction(camera.position, point_on_screen);
    Ray ray(camera.position, direction);

    img[tidY*width + tidX] = cast_ray(camera.position, ray, spheres, spheres_count);
}

Image render(std::vector<Sphere> const &spheres, Camera &camera) {
    hipSetDevice(2);
    Image img(1920, 1080);

    RGBColor *cudaPixels;
    gpuErrchk(hipMalloc(&cudaPixels, sizeof(RGBColor) * img.width() * img.height()));

    Sphere *cudaSpheres;
    gpuErrchk(hipMalloc(&cudaSpheres, sizeof(Sphere) * spheres.size()));
    gpuErrchk(hipMemcpy(cudaSpheres, spheres.data(), sizeof(Sphere) * spheres.size(), hipMemcpyHostToDevice));

    uint32_t gridX = (img.width() + CHUNK_SIZE - 1) / CHUNK_SIZE;
    uint32_t gridY = (img.height() + CHUNK_SIZE - 1) / CHUNK_SIZE;

    kernel<<<{gridX, gridY}, {CHUNK_SIZE, CHUNK_SIZE}>>>(img.width(), img.height(), cudaPixels, cudaSpheres, camera,
                                                         spheres.size());

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(img.raw(), cudaPixels, sizeof(RGBColor) * img.width() * img.height(), hipMemcpyDeviceToHost));
    hipFree(cudaPixels);
    hipFree(cudaSpheres);

    return img;
}
