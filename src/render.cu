#include "hip/hip_runtime.h"
#include <limits>
#include <vector>
#include "camera.h"
#include "color.h"
#include "ray.h"
#include "rectangle.h"
#include "render.h"
#include "vector3.h"

#define BACKGROUND_COLOR RGBColor(0, 1, 1);
#define FLOOR_COLOR RGBColor(0, 1, 0);

static constexpr int CHUNK_SIZE = 32;
static constexpr int MAX_NUMBER_OF_REFLECTIONS = 100;
static constexpr float FLOAT_INFINITY = std::numeric_limits<float>::infinity();

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort) exit(code);
    }
}

static __device__ RGBColor cast_ray(Ray ray, Sphere *spheres,
                                    int const &spheres_count) {
    int hit_sphere = -1;
    bool was_hit;
    float color_multiplier = 1;
    RGBColor color(0,0,0);
    for (int i = 0; i < MAX_NUMBER_OF_REFLECTIONS; i++) {
        float current_distance = FLOAT_INFINITY;
        was_hit = false;

        for (int sphere_index = 0; sphere_index < spheres_count;
             sphere_index++) {
            if (sphere_index != hit_sphere &&
                spheres[sphere_index].hits_ray(ray)) {
                was_hit = true;
                float distance =
                    Vector3(ray.get_position(),
                            spheres[sphere_index].get_intersection_point(ray))
                        .length();
                if (distance < current_distance) {
                    current_distance = distance;
                    hit_sphere = sphere_index;
                }
            }
        }

        if (was_hit) {
            color = color + color_multiplier * spheres[hit_sphere].get_material().get_color();
            color_multiplier *= spheres[hit_sphere].get_material().get_reflection_coefficient();
            ray = spheres[hit_sphere].reflect(ray);
            if(color_multiplier < 0.001f) {
                break;
            }
        } else {
            break;
        }
    }

    if (ray.get_direction().y < 0) {
        color = color + color_multiplier * FLOOR_COLOR;
    } else {
        color = color + color_multiplier * BACKGROUND_COLOR;
    }

    return color;
}

static __global__ void kernel(int width, int height, RGBColor *img,
                              Sphere *spheres, Camera camera,
                              int spheres_count) {
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    if (tidX > width || tidY > height) {
        return;
    }

    img[tidY * width + tidX] = RGBColor(0.0f, 0.0f, 0.0f);

    Rectangle screen = camera.get_screen();

    Vector3 point_on_screen =
        Vector3(screen.left_top_point.x + tidX * screen.width() / width,
                screen.left_top_point.y - tidY * screen.height() / height,
                screen.left_top_point.z);
    Vector3 direction(camera.position, point_on_screen);
    Ray ray(camera.position, direction);

    img[tidY * width + tidX] = cast_ray(ray, spheres, spheres_count);
}

Image render(std::vector<Sphere> const &spheres, Camera &camera) {
    hipSetDevice(0);
    Image img(1920, 1080);

    RGBColor *cudaPixels;
    gpuErrchk(
        hipMalloc(&cudaPixels, sizeof(RGBColor) * img.width() * img.height()));

    Sphere *cudaSpheres;
    gpuErrchk(hipMalloc(&cudaSpheres, sizeof(Sphere) * spheres.size()));
    gpuErrchk(hipMemcpy(cudaSpheres, spheres.data(),
                         sizeof(Sphere) * spheres.size(),
                         hipMemcpyHostToDevice));

    uint32_t gridX = (img.width() + CHUNK_SIZE - 1) / CHUNK_SIZE;
    uint32_t gridY = (img.height() + CHUNK_SIZE - 1) / CHUNK_SIZE;

    kernel<<<{gridX, gridY}, {CHUNK_SIZE, CHUNK_SIZE}>>>(
        img.width(), img.height(), cudaPixels, cudaSpheres, camera,
        spheres.size());

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(img.raw(), cudaPixels,
                         sizeof(RGBColor) * img.width() * img.height(),
                         hipMemcpyDeviceToHost));
    hipFree(cudaPixels);
    hipFree(cudaSpheres);

    return img;
}
